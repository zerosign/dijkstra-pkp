#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "defs.h"
#include "reader.h"



__device__ int index() {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int x(int index) {
	return index;
}

__device__ int y(int index) {
	return index;
}

__global__ void first_cuda_ssp_kernel() {
	
}	

__global__ void second_cuda_ssp_kernel() {
	
}

<<<<<<< HEAD
=======
/**
void AdjMapToMatrix(g::adjmap & graph, 
	float * VertexArray,
	float * WeightArray);
**/
>>>>>>> origin/zerosign

int main(int argc, char ** argv) {
	
	if(argc != 4) {
		std::fprintf(stderr, "\n[Usage] : %s [filename]\n", 
			std::string(argv[0]).c_str());
	}

	const char * filename = argv[1];

	//g::adjmap graph;

	//g::reader::read(std::string(filename).c_str(), graph);

	int * VertexArray, * EdgeArray, * MaskArray;
	float * WeightArray, * CostArray, * UpdateCostArray;
<<<<<<< HEAD
	bool * MaskArray;

	hipEvent_t start, stop;
=======
	
	VertexArray = malloc(sizeof(

	AdjMapToMatrix(graph, VertexArray, WeightArray);

>>>>>>> origin/zerosign

	return 0;
}


