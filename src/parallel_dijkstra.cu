#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <list>
#include <sstream>
#include <iterator>
#include <iostream>
#include "defs.h"
#include "file.h"

__device__ int index() {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void first_cuda_ssp_kernel(float * WeightArray, 
		int * MaskArray, float * CostArray, 
		float * UpdateCostArray) {
	
	//int id = index();
	
	if(MaskArray[blockIdx.x] == 1) {
		MaskArray[blockIdx.x] = 0;
		
		//int vertex = threadIdx.x;
		//int neighborSize = blockDim.x;

		for(int ii = 0; ii < blockDim.x; ii++) {
			
			if(threadIdx.x == ii) 
				continue;
			
			if(UpdateCostArray[threadIdx.x] > CostArray[threadIdx.x] + WeightArray[index()]) {
				UpdateCostArray[threadIdx.x] = CostArray[threadIdx.x] + WeightArray[index()];
			}
		}

	}
}	


__global__ void second_cuda_ssp_kernel(float * WeightArray,
		int * MaskArray, float * CostArray, 
		float * UpdateCostArray) {

	// Update the cost array
	if(CostArray[threadIdx.x] > UpdateCostArray[threadIdx.x]) {
		CostArray[threadIdx.x] = UpdateCostArray[threadIdx.x];
		MaskArray[threadIdx.x] = 1;
		//VertexArray[blockIdx.x] = threadIdx.x;
	}
	UpdateCostArray[threadIdx.x] = CostArray[threadIdx.x];
}

bool is_empty(int * MaskArrayHost, int size) {
	bool not_empty = false;
	for(int ii = 0; ii < size; ii++) {
		not_empty |= (MaskArrayHost[ii] == 1) ? (true) : (false);
		if(not_empty)
			return false;
	}
	return !not_empty;
}

std::list<int> get_shortest_path(float * CostArray, 
		const int source, const int target,
		const int vertexSize,
		int & finalCost);

int find_min_index(float * Cost, int N) {
	int index = -1;
	int value = std::numeric_limits<int>::max();
	for(int ii = 0; ii < N; ii++) {
		if(value > Cost[ii]) {
			index = ii;
			value = Cost[ii];
		}
	}
	if(index == -1){
		std::fprintf(stderr, "[Error] Error nggak jelas\n");
		exit(EXIT_FAILURE);
	}
	return index;
}

int main(int argc, char ** argv) {
	
	if(argc != 4) {
		std::fprintf(stderr, "\n[Usage] : %s [filename] [start] [end] [outfile]\n", 
			std::string(argv[0]).c_str());
		exit(EXIT_FAILURE);
	}

	const char * filename = argv[1];

	int start, end;

	start = std::atoi(argv[2]);
	end = std::atoi(argv[3]);


	dim3 gridDim, blockDim;
	//g::adjmap graph;

	//g::reader::read(std::string(filename).c_str(), graph);


	// untuk MaskArray sizenya adalah size of vertex (dilihat dari distance yang ingin diubah)
	//int * VertexArrayHost, * VertexArrayDevice, 
	int * MaskArrayHost, * MaskArrayDevice;

	float * WeightArrayHost, * WeightArrayDevice,
			* CostArrayHost, * CostArrayDevice, 
			* UpdateCostArrayHost, * UpdateCostArrayDevice;

	int vertexSize = 0;


	io::file::read(filename, vertexSize, WeightArrayHost);

	if(vertexSize < 2) {
		std::fprintf(stderr, "\n[ERROR] Anomaly in VertexSize\n");
		exit(EXIT_FAILURE);
	}


	// RAW copy from vertex matrix to block
	gridDim.x = vertexSize;
	blockDim.x = vertexSize;
	
	/**
	for(int ii = 0; ii < vertexSize; ii++) {
		for(int jj = 0; jj < vertexSize; jj++) {
			std::printf("%f ", WeightArrayHost[ii * vertexSize + jj]);
		}
		std::printf("\n");
	}
	**/

	const int rawVertexSize = vertexSize * sizeof(float);
	const int rawMatrixSize = rawVertexSize * rawVertexSize;

	// Cost array is used for counting
	// cost of given source to target
	CostArrayHost = (float*)malloc(rawVertexSize);

	// temporary cost array for counting 
	// cost from the previous vertex to current 
	// vertex
	// 
	// If it's smaller than CostArray then it's switched
	UpdateCostArrayHost = (float*)malloc(rawVertexSize);	
	
	//VertexArrayHost = (float*) malloc(rawVertexSize);

	MaskArrayHost = (int*)malloc(rawVertexSize);

	for(int ii = 0; ii < vertexSize; ii++) {
		CostArrayHost[ii] = std::numeric_limits<int>::max();
		UpdateCostArrayHost[ii] = std::numeric_limits<int>::max();
	}
	
	//hipMalloc((void**)&VertexArrayDevice, rawVertexSize);
	hipMalloc((void**)&WeightArrayDevice, rawMatrixSize);
	hipMalloc((void**)&CostArrayDevice, rawVertexSize);
	hipMalloc((void**)&UpdateCostArrayDevice, rawVertexSize);
	
	// MaskArray is used for determining that the vertex
	// is already been visited or not
	hipMalloc((void**)&MaskArrayDevice, rawVertexSize);
	
	MaskArrayHost[start] = 1;
	CostArrayHost[start] = 0;
	UpdateCostArrayHost[start] = 0;
	//VertexArrayHost[0] = start;


	//hipMemcpy(VertexArrayDevice, VertexArrayHost, 
	//		rawVertexSize, hipMemcpyHostToDevice);
	
	hipMemcpy(WeightArrayDevice, WeightArrayHost,
			rawMatrixSize, hipMemcpyHostToDevice);

	hipMemcpy(CostArrayDevice, CostArrayHost,
			rawVertexSize, hipMemcpyHostToDevice);
	
	hipMemcpy(UpdateCostArrayDevice, UpdateCostArrayHost,
			rawVertexSize, hipMemcpyHostToDevice);
	
	// freeing big memory that not been used
	// anymore because already copied to device
	free(UpdateCostArrayHost);
	free(WeightArrayHost);
	//free(VertexArrayHost);
	
	std::vector<int> path;
	float finalCost = 0;
	
	int counter = 0;

	while(!is_empty(MaskArrayHost, vertexSize * vertexSize)) {
		for(int ii = 0; ii < vertexSize; ii++) {
			
			first_cuda_ssp_kernel<<<gridDim, blockDim >>>(WeightArrayDevice,
					MaskArrayDevice, CostArrayDevice, UpdateCostArrayDevice);

			second_cuda_ssp_kernel<<<gridDim, blockDim >>>(WeightArrayDevice, 
					MaskArrayDevice, CostArrayDevice, UpdateCostArrayDevice);
		}

		// update the masks
		hipMemcpy(MaskArrayHost, MaskArrayDevice, 
				rawVertexSize, hipMemcpyDeviceToHost);
		
		// get the cost of the current vertex
		hipMemcpy(CostArrayHost, CostArrayDevice,
				rawVertexSize, hipMemcpyDeviceToHost);

		// find minimum of each vertex
		int index = find_min_index(CostArrayHost, vertexSize);
		path.push_back(index);
		finalCost += CostArrayHost[index];
	
		std::printf("Counter : %d\n", ++counter);
	}
	
	

	std::stringstream sstream(std::stringstream::in |
			std::stringstream::out);

	std::copy(path.begin(), path.end(), 
			std::ostream_iterator<g::vertex_t>(sstream, "->"));	
	
	std::string out;

	//while(!sstream.eof()) 
	//	sstream >> out;
	out = sstream.str();
	
	std::cout << "Cost : \n" << finalCost << std::endl;
	std::cout << "Path : \n" << out << "end" << std::endl;
	
	return 0;
}


std::list<int> get_shortest_path(float * CostArray, 
		const int source, const int target,
		const int vertexSize, int & finalCost) {
	
	std::list<int> result;
	
	
	return result;
}
