#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "defs.h"
#include "file.h"


__device__ int index() {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int x(int index) {
	return index;
}

__device__ int y(int index) {
	return index;
}

__global__ void first_cuda_ssp_kernel(float * VertexArray,
		float * WeightArray, float * MaskArray, float * CostArray, 
		float * UpdateCostArray) {
	
	int id = index();
	
	if(MaskArray[id] == 1) {
		MaskArray[id] = 0;
		
		int vertex = x(index);
		int neighborSize = y(N);

		for(int ii = 0; ii < neighborSize; ii++) {
			
			if(y(id) == ii) 
				continue;

			int nid = index * neighborSize + ii;
			
			if(UpdateCostArray[nid] > CostArray[id] + WeightArray[nid]) {
				UpdateCostArray[nid] = CostArray[id] + WeightArray[nid];
			}
		}

	}

}	

__global__ void second_cuda_ssp_kernel(float * VertexArray,
		float * WeightArray, float * MaskArray, float * CostArray,
		float * UpdateCostArray) {
	int id = index();

	if(CostArray[id] > UpdateCostArray[id]) {
		CostArray[id] = UpdateCostArray[id];
		MaskArray[id] = 1;
	}
	UpdateCostArray[id] = CostArray[id];
}

bool is_empty(int * MaskArrayHost, int size) {
	bool not_empty = false;
	for(int ii = 0; ii < size; ii++) {
		not_empty |= (MaskArrayHost[ii] == 1) ? (true) : (false);
		if(not_empty)
			return false;
	}
	return !not_empty;
}

int main(int argc, char ** argv) {
	
	if(argc != 4) {
		std::fprintf(stderr, "\n[Usage] : %s [filename] [start] [end] [outfile]\n", 
			std::string(argv[0]).c_str());
		exit(EXIT_FAILURE);
	}

	const char * filename = argv[1];

	int start, end;

	start = std::atoi(argv[2]);
	end = std::atoi(argv[3]);


	dim3 gridDim, blockDim;
	//g::adjmap graph;

	//g::reader::read(std::string(filename).c_str(), graph);

	int * VertexArrayHost, * VertexArrayDevice, 
		 * MaskArrayHost, * MaskArrayDevice;

	float * WeightArrayHost, * WeightArrayDevice,
			* CostArrayHost, * CostArrayDevice, 
			* UpdateCostArrayHost, * UpdateCostArrayDevice;

	int vertexSize = 1;

	io::file::read(filename, vertexSize,
			VertexArrayHost, WeightArrayHost);

	gridDim.x = vertexSize;
	blockDim.x = vertexSize;

	CostArrayHost = (float*)malloc(vertexSize * sizeof(float));
	UpdateCostArrayHost = (float*)malloc(vertexSize * sizeof(float));	
	
	for(int ii = 0; ii < vertexSize; ii++) {
		CostArrayHost[ii] = std::numeric_limits<int>::max();
		UpdateCostArrayHost[ii] = std::numeric_limits<int>::max();
	}

	
	hipMalloc((void**)&VertexArrayDevice, vertexSize * sizeof(float));
	hipMalloc((void**)&WeightArrayDevice, vertexSize * vertexSize * sizeof(float) * sizeof(float));
	hipMalloc((void**)&CostArrayDevice, vertexSize * sizeof(float));
	hipMalloc((void**)&UpdateCostArrayDevice, vertexSize * sizeof(float));
	
	// malloc default set it to zero (we call this as a false)
	hipMalloc((void**)&MaskArray, vertexSize * sizeof(float));
	
	MaskArrayHost[start] = 1;
	CostArrayHost[start] = 0;
	UpdateCostArrayHost[start] = 0;

	hipMemcpy(VertexArrayDevice, VertexArrayHost, 
			size, hipMemcpyHostToDevice);
	
	hipMemcpy(WeightArrayDevice, WeightArrayHost,
			size, hipMemcpyHostToDevice);

	hipMemcpy(CostArrayDevice, CostArrayHost,
			size, hipMemcpyHostToDevice);
	
	hipMemcpy(UpdateCostArrayDevice, UpdateCostArrayHost,
			size, hipMemcpyHostToDevice);

	while(!is_empty(MaskArrayHost, vertexSize * vertexSize)) {
		for(int ii = 0; ii < vertexSize; ii++) {
			
			first_cuda_ssp_kernel<<<gridDim, blockDim >>>(VertexArrayDevice, 
					WeightArrayDevice, MaskArrayDevice, CostArrayDevice,
					UpdateCostArrayDevice);

			second_cuda_ssp_kernel<<<gridDim, blockDim >>>(VertexArrayDevice,
					WeightArrayDevice, MaskArrayDevice, CostArrayDevice,
					UpdateCostArrayDevice);
		}

		// update the masks
		hipMemcpy(MaskArrayHost, MaskArrayDevice, 
				size, hipMemcpyDeviceToHost);
	}

	return 0;
}


