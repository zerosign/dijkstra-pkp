#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "defs.h"
#include "file.h"


__device__ int index() {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int x(int index) {
	return index;
}

__device__ int y(int index) {
	return index;
}

__global__ void first_cuda_ssp_kernel(int * VertexArray,
		float * WeightArray, int * MaskArray, float * CostArray, 
		float * UpdateCostArray) {
	
	int id = index();
	
	if(MaskArray[id] == 1) {
		MaskArray[id] = 0;
		
		int vertex = x(index);
		int neighborSize = y(blockDim.x);

		for(int ii = 0; ii < neighborSize; ii++) {
			
			if(y(id) == ii) 
				continue;

			int nid = index * neighborSize + ii;
			
			if(UpdateCostArray[nid] > CostArray[id] + WeightArray[nid]) {
				UpdateCostArray[nid] = CostArray[id] + WeightArray[nid];
			}
		}

	}

}	

__global__ void second_cuda_ssp_kernel(int * VertexArray,
		float * WeightArray, int * MaskArray, float * CostArray,
		float * UpdateCostArray) {
	int id = index();

	if(CostArray[id] > UpdateCostArray[id]) {
		CostArray[id] = UpdateCostArray[id];
		MaskArray[id] = 1;
	}
	UpdateCostArray[id] = CostArray[id];
}

bool is_empty(int * MaskArrayHost, int size) {
	bool not_empty = false;
	for(int ii = 0; ii < size; ii++) {
		not_empty |= (MaskArrayHost[ii] == 1) ? (true) : (false);
		if(not_empty)
			return false;
	}
	return !not_empty;
}

int main(int argc, char ** argv) {
	
	if(argc != 4) {
		std::fprintf(stderr, "\n[Usage] : %s [filename] [start] [end] [outfile]\n", 
			std::string(argv[0]).c_str());
		exit(EXIT_FAILURE);
	}

	const char * filename = argv[1];

	int start, end;

	start = std::atoi(argv[2]);
	end = std::atoi(argv[3]);


	dim3 gridDim, blockDim;
	//g::adjmap graph;

	//g::reader::read(std::string(filename).c_str(), graph);

	int * VertexArrayHost, * VertexArrayDevice, 
		 * MaskArrayHost, * MaskArrayDevice;

	float * WeightArrayHost, * WeightArrayDevice,
			* CostArrayHost, * CostArrayDevice, 
			* UpdateCostArrayHost, * UpdateCostArrayDevice;

	int vertexSize = 1;

	io::file::read(filename, vertexSize,
			VertexArrayHost, WeightArrayHost);

	gridDim.x = vertexSize;
	blockDim.x = vertexSize;

	const int rawVertexSize = vertexSize * sizeof(float);
	const int rawMatrixSize = rawVertexSize * rawVertexSize;

	CostArrayHost = (float*)malloc(rawVertexSize);
	UpdateCostArrayHost = (float*)malloc(rawVertexSize);	
	
	for(int ii = 0; ii < vertexSize; ii++) {
		CostArrayHost[ii] = std::numeric_limits<int>::max();
		UpdateCostArrayHost[ii] = std::numeric_limits<int>::max();
	}
	
	hipMalloc((void**)&VertexArrayDevice, rawVertexSize);
	hipMalloc((void**)&WeightArrayDevice, rawMatrixSize);
	hipMalloc((void**)&CostArrayDevice, rawVertexSize);
	hipMalloc((void**)&UpdateCostArrayDevice, rawVertexSize);
	
	// malloc default set it to zero (we call this as a false)
	hipMalloc((void**)&MaskArrayDevice, rawVertexSize);
	
	MaskArrayHost[start] = 1;
	CostArrayHost[start] = 0;
	UpdateCostArrayHost[start] = 0;

	hipMemcpy(VertexArrayDevice, VertexArrayHost, 
			rawVertexSize, hipMemcpyHostToDevice);
	
	hipMemcpy(WeightArrayDevice, WeightArrayHost,
			rawMatrixSize, hipMemcpyHostToDevice);

	hipMemcpy(CostArrayDevice, CostArrayHost,
			rawVertexSize, hipMemcpyHostToDevice);
	
	hipMemcpy(UpdateCostArrayDevice, UpdateCostArrayHost,
			rawVertexSize, hipMemcpyHostToDevice);

	while(!is_empty(MaskArrayHost, vertexSize * vertexSize)) {
		for(int ii = 0; ii < vertexSize; ii++) {
			
			first_cuda_ssp_kernel<<<gridDim, blockDim >>>(VertexArrayDevice, 
					WeightArrayDevice, MaskArrayDevice, CostArrayDevice,
					UpdateCostArrayDevice);

			second_cuda_ssp_kernel<<<gridDim, blockDim >>>(VertexArrayDevice,
					WeightArrayDevice, MaskArrayDevice, CostArrayDevice,
					UpdateCostArrayDevice);
		}

		// update the masks
		hipMemcpy(MaskArrayHost, MaskArrayDevice, 
				rawVertexSize, hipMemcpyDeviceToHost);
	}

	return 0;
}


