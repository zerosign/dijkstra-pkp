#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "defs.h"
#include "file.h"


__device__ int index() {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int x(int index) {
	return index;
}

__device__ int y(int index) {
	return index;
}

__global__ void first_cuda_ssp_kernel(float * VertexArray,
		float * EdgeArray, float * WeightArray,
		float * MaskArray, float * CostArray, 
		float * UpdateCostArray) {
	
}	

__global__ void second_cuda_ssp_kernel() {
	
}


int main(int argc, char ** argv) {
	
	if(argc != 4) {
		std::fprintf(stderr, "\n[Usage] : %s [filename] [start] [end] [outfile]\n", 
			std::string(argv[0]).c_str());
	}

	const char * filename = argv[1];

	int start, end;

	start = std::atoi(argv[2]);
	end = std::atoi(argv[3]);

	//g::adjmap graph;

	//g::reader::read(std::string(filename).c_str(), graph);

	int * VertexArrayHost, * VertexArrayDevice,
		 * EdgeArray, * MaskArray;

	float * WeightArrayHost, * WeightArrayDevice,
			* CostArrayHost, * CostArrayDevice, 
			* UpdateCostArrayHost, * UpdateCostArrayDevice;

	int vertexSize;

	io::file::read(std::string(filename).c_str(), vertexSize
			VertexArrayHost, WeightArrayHost);

	CostArrayHost = (int*)malloc(vertexSize);
	UpdateCostArrayHost = (int*)malloc(vertexSize);
	
	
	for(int ii = 0; ii < vertexSize; ii++) {
		CostArrayHost[ii] = std::numeric_limits<int>::max();
		UpdateCostArrayHost[ii] = std::numeric_limits<int>::max();
	}

	
	hipMalloc((void**)&VertexArrayDevice, vertexSize);
	hipMalloc((void**)&WeightArrayDevice, vertexSize * vertexSize);
	hipMalloc((void**)&CostArrayDevice, vertexSize);
	hipMalloc((void**)&UpdateCostArrayDevice, vertexSize);
	
	// malloc default set it to zero (we call this as a false)
	hipMalloc((void**)&MaskArray, vertexSize);
	
	/**
	MaskArrayHost[start] = 1;
	CostArrayHost[start] = 0;
	UpdateCostArrayHost[start] = 0;

	hipMemcpy(VertexArrayDevice, VertexArrayHost, 
			size, hipMemcpyHostToDevice);
	
	hipMemcpy(WeightArrayDevice, WeightArrayHost,
			size, hipMemcpyHostToDevice);

	hipMemcpy(CostArrayDevice, CostArrayHost,
			size, hipMemcpyHostToDevice);
	
	hipMemcpy(UpdateCostArrayDevice, UpdateCostArrayHost,
			size, hipMemcpyHostToDevice);

	while(true) {
	
	}
	**/
	return 0;
}


