#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "defs.h"
#include "reader.h"




__device__ int index() {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int x(int index) {
	return index;
}

__device__ int y(int index) {
	return index;
}

__global__ void first_cuda_ssp_kernel(float * VertexArray,
		float * EdgeArray, float * WeightArray,
		float * MaskArray, float * CostArray, 
		float * UpdateCostArray) {
		
	
}	

__global__ void second_cuda_ssp_kernel() {
	
}


int main(int argc, char ** argv) {
	
	if(argc != 4) {
		std::fprintf(stderr, "\n[Usage] : %s [filename]\n", 
			std::string(argv[0]).c_str());
	}

	const char * filename = argv[1];

	int start, end;

	start = std::atoi(argv[2]);
	end = std::atoi(argv[3]);

	//g::adjmap graph;

	//g::reader::read(std::string(filename).c_str(), graph);

	int * VertexArrayHost, * VertexArrayDevice,
		 * EdgeArray, * MaskArray;
	float * WeightArrayHost, * WeightArrayDevice,
			* CostArrayHost, * CostArrayDevice, 
			* UpdateCostArrayHost, * UpdateCostArrayDevice;

	io::file::read(std::string(filename).c_str(),
			VertexArrayHost, WeightArrayHost);
	
	CostArrayHost = (int*)malloc(size);
	UpdateCostArrayHost = (int*)malloc(size);

	for(int ii = 0; ii < size; ii++) {
		CostArrayHost[ii] = std::limits<int>::max();
		UpdateCostArrayHost[ii] = std::limits<int>::max()t
	}

	hipMalloc((void**)&VertexArrayDevice, size);
	hipMalloc((void**)&WeightArrayDevice, size);
	hipMalloc((void**)&CostArrayDevice, size);
	hipMalloc((void**)&UpdateCostArrayDevice, size);
	
	// malloc default set it to zero (we call this as a false)
	hipMalloc((void**)&MaskArray, size);
	
	MaskArrayHost[start] = 1;
	CostArrayHost[start] = 0;
	UpdateCostArrayHost[start] = 0;

	hipMemcpy(VertexArrayDevice, VertexArrayHost, 
			size, hipMemcpyHostToDevice);
	
	hipMemcpy(WeightArrayDevice, WeightArrayHost,
			size, hipMemcpyHostToDevice);

	hipMemcpy(CostArrayDevice, CostArrayHost,
			size, hipMemcpyHostToDevice);
	
	hipMemcpy(UpdateCostArrayDevice, UpdateCostArrayHost,
			size, hipMemcpyHostToDevice);


	while(true) {
	}

	return 0;
}


