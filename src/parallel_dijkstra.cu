#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "defs.h"
#include "file.h"


__device__ int index() {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int x(int index) {
	return index;
}

__device__ int y(int index) {
	return index;
}

__global__ void first_cuda_ssp_kernel(float * VertexArray,
		float * EdgeArray, float * WeightArray,
		float * MaskArray, float * CostArray, 
		float * UpdateCostArray) {
	
}	

__global__ void second_cuda_ssp_kernel() {
	
}


int main(int argc, char ** argv) {
	
	if(argc != 4) {
		std::fprintf(stderr, "\n[Usage] : %s [filename] [start] [end] [outfile]\n", 
			std::string(argv[0]).c_str());
		exit(EXIT_FAILURE);
	}

	const char * filename = argv[1];

	int start, end;

	start = std::atoi(argv[2]);
	end = std::atoi(argv[3]);

	//g::adjmap graph;

	//g::reader::read(std::string(filename).c_str(), graph);

	int * VertexArrayHost, * VertexArrayDevice,
		 * EdgeArray, * MaskArray;

	float * WeightArrayHost, * WeightArrayDevice,
			* CostArrayHost, * CostArrayDevice, 
			* UpdateCostArrayHost, * UpdateCostArrayDevice;

	int vertexSize = 1;

	io::file::read(filename, vertexSize,
			VertexArrayHost, WeightArrayHost);

	CostArrayHost = (float*)malloc(vertexSize * sizeof(float));
	UpdateCostArrayHost = (float*)malloc(vertexSize * sizeof(float));
	
	
	for(int ii = 0; ii < vertexSize; ii++) {
		CostArrayHost[ii] = std::numeric_limits<int>::max();
		UpdateCostArrayHost[ii] = std::numeric_limits<int>::max();
	}

	
	hipMalloc((void**)&VertexArrayDevice, vertexSize * sizeof(float));
	hipMalloc((void**)&WeightArrayDevice, vertexSize * vertexSize * sizeof(float) * sizeof(float));
	hipMalloc((void**)&CostArrayDevice, vertexSize * sizeof(float));
	hipMalloc((void**)&UpdateCostArrayDevice, vertexSize * sizeof(float));
	
	// malloc default set it to zero (we call this as a false)
	hipMalloc((void**)&MaskArray, vertexSize * sizeof(float));
	
	/**
	MaskArrayHost[start] = 1;
	CostArrayHost[start] = 0;
	UpdateCostArrayHost[start] = 0;

	hipMemcpy(VertexArrayDevice, VertexArrayHost, 
			size, hipMemcpyHostToDevice);
	
	hipMemcpy(WeightArrayDevice, WeightArrayHost,
			size, hipMemcpyHostToDevice);

	hipMemcpy(CostArrayDevice, CostArrayHost,
			size, hipMemcpyHostToDevice);
	
	hipMemcpy(UpdateCostArrayDevice, UpdateCostArrayHost,
			size, hipMemcpyHostToDevice);

	while(true) {
	
	}
	**/
	return 0;
}


