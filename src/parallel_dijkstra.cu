#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "defs.h"
#include "reader.h"



__device__ int index() {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int x(int index) {
	return index;
}

__device__ int y(int index) {
	return index;
}

__global__ void first_cuda_ssp_kernel() {
	
}

__global__ void second_cuda_ssp_kernel() {

}

/**
void AdjMapToMatrix(g::adjmap & graph, 
	float * VertexArray,
	float * WeightArray);
**/

int main(int argc, char ** argv) {
	
	if(argc != 4) {
		std::fprintf(stderr, "\n[Usage] : %s [filename]\n", 
			std::string(argv[0]).c_str());
	}

	const char * filename = argv[1];

	//g::adjmap graph;

	//g::reader::read(std::string(filename).c_str(), graph);

	int * VertexArray, * EdgeArray, * MaskArray;
	float * WeightArray, * CostArray, * UpdateCostArray;
	
	VertexArray = malloc(sizeof(

	AdjMapToMatrix(graph, VertexArray, WeightArray);


	return 0;
}


