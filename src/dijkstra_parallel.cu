#include <cstdio>
#include <hip/hip_runtime.h>
#include <list>
#include "file.h"


#define BUFFSIZE 100

typedef struct Memory {
	float * Host, * Device;
	operator()(int N) {
		Host = (float*)malloc(sizeof(N));
		hipMalloc((void**)&Device, N);
	}
	operator~() {
		free(Host);
		hipFree(Device);
	}
};

void read(const char * filename, int & vertexSize,
	struct Memory& weights) {
		
	std::ifstream input(std::string(filename).c_str());
	
	if(!input.good()) {
		std::fprintf(stderr, "[ERROR] No such file %s\n", 
				std::string(filename).c_str());
		exit(EXIT_FAILURE);
	}
	char * buffer = new char[BUFFSIZE];

	input.getline(buffer, BUFFSIZE);
		
	//int num_vertices, num_edges;

	int edgeSize;

	std::sscanf(buffer, "%d\t%d\n", &vertexSize,
			&edgeSize);


	//VertexArray = (int*)malloc(sizeof(int)*num_vertices);	
	//WeightArray = (float*)malloc(sizeof(float)*(num_vertices*num_vertices));
	
	weights = Memory(vertexSize * vertexSize * sizeof(float));

	//for(int ii = 0; ii < num_vertices; ii++) {
	//	VertexArray[ii] = ii;
	//}

	for(int ii = 0; ii < edgeSize; ii++) {
		int start, end;
		float weight;
		//sstream >> start >> end >> weight;
		buffer = new char[BUFFSIZE];

		input.getline(buffer, BUFFSIZE);
	
		std::sscanf(buffer, "%d\t%d\t%f\n", &start,
				&end, &weight);

		weights.Host[start * vertexSize + end ] = weight;
		weights.Host[end * vertexSize + start] = weight;
	}
	
	input.close();
	
}



int main(int argc, char ** argv) {
	
	const char * filename = argv[1];

	int start, end, N;

	start = std::atoi(argv[2]);
	end = std::atoi(argv[3]);

	dim3 gridDim, blockDim;

	struct Memory WeightArray;
	
	io::file::read(filename, N, WeightArray);
	
	gridDim.x = N;
	blockDim.x = N;

	for(int ii = 0; ii < N; ii++) {
		for(int jj = 0; jj < N; jj++) {
			std::printf("%f ", WeightArray.Host[ii * vertexSize + jj]);
		}
		std::printf("\n");
	}



	return 0;
}
