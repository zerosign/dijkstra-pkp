#include "hip/hip_runtime.h"
#include <cstdio>
//#include <hip/hip_runtime.h>
#include <cstdlib>
#include <list>
#include <sstream>
#include <iterator>
#include <iostream>
#include "defs.h"
#include "file.h"

__device__ int index() {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int x(int index) {
	return index;
}

__device__ int y(int index) {
	return index;
}

__global__ void first_cuda_ssp_kernel(int * VertexArray,
		float * WeightArray, int * MaskArray, float * CostArray, 
		float * UpdateCostArray) {
	
	int id = index();
	
	if(MaskArray[id] == 1) {
		MaskArray[id] = 0;
		
		int vertex = x(id);
		int neighborSize = y(blockDim.x);

		for(int ii = 0; ii < neighborSize; ii++) {
			
			if(y(id) == ii) 
				continue;

			int nid = id * neighborSize + ii;
			
			if(UpdateCostArray[nid] > CostArray[id] + WeightArray[nid]) {
				UpdateCostArray[nid] = CostArray[id] + WeightArray[nid];
			}
		}

	}

}	



__global__ void second_cuda_ssp_kernel(int * VertexArray,
		float * WeightArray, int * MaskArray, float * CostArray,
		float * UpdateCostArray) {
	int id = index();

	// Update the cost array
	if(CostArray[id] > UpdateCostArray[id]) {
		CostArray[id] = UpdateCostArray[id];
		MaskArray[id] = 1;
	}
	UpdateCostArray[id] = CostArray[id];
}

bool is_empty(int * MaskArrayHost, int size) {
	bool not_empty = false;
	for(int ii = 0; ii < size; ii++) {
		not_empty |= (MaskArrayHost[ii] == 1) ? (true) : (false);
		if(not_empty)
			return false;
	}
	return !not_empty;
}

std::list<int> get_shortest_path(float * CostArray, 
		const int source, const int target,
		const int vertexSize,
		int & finalCost);

int main(int argc, char ** argv) {
	
	if(argc != 4) {
		std::fprintf(stderr, "\n[Usage] : %s [filename] [start] [end] [outfile]\n", 
			std::string(argv[0]).c_str());
		exit(EXIT_FAILURE);
	}

	const char * filename = argv[1];

	int start, end;

	start = std::atoi(argv[2]);
	end = std::atoi(argv[3]);


	dim3 gridDim, blockDim;
	//g::adjmap graph;

	//g::reader::read(std::string(filename).c_str(), graph);


	// untuk VertexArray sizenya adalah size of vertex
	// untuk MaskArray sizenya adalah size of vertex (dilihat dari distance yang ingin diubah)
	int * VertexArrayHost, * VertexArrayDevice, 
		 * MaskArrayHost, * MaskArrayDevice;

	float * WeightArrayHost, * WeightArrayDevice,
			* CostArrayHost, * CostArrayDevice, 
			* UpdateCostArrayHost, * UpdateCostArrayDevice;

	int vertexSize = 0;

	io::file::read(filename, vertexSize,
			VertexArrayHost, WeightArrayHost);

	if(vertexSize < 2) {
		std::fprintf(stderr, "\n[ERROR] Anomaly in VertexSize\n");
		exit(EXIT_FAILURE);
	}


	// RAW copy from vertex matrix to block
	gridDim.x = vertexSize;
	blockDim.x = vertexSize;
	
	/**
	for(int ii = 0; ii < vertexSize; ii++) {
		for(int jj = 0; jj < vertexSize; jj++) {
			std::printf("%f ", WeightArrayHost[ii * vertexSize + jj]);
		}
		std::printf("\n");
	}
	**/

	const int rawVertexSize = vertexSize * sizeof(float);
	const int rawMatrixSize = rawVertexSize * rawVertexSize;

	// Cost array is used for counting
	// cost of given source to target
	CostArrayHost = (float*)malloc(rawVertexSize);

	// temporary cost array for counting 
	// cost from the previous vertex to current 
	// vertex
	// 
	// If it's smaller than CostArray then it's switched
	UpdateCostArrayHost = (float*)malloc(rawVertexSize);	
	
	MaskArrayHost = (int*)malloc(rawVertexSize);

	for(int ii = 0; ii < vertexSize; ii++) {
		CostArrayHost[ii] = std::numeric_limits<int>::max();
		UpdateCostArrayHost[ii] = std::numeric_limits<int>::max();
	}
	
	hipMalloc((void**)&VertexArrayDevice, rawVertexSize);
	hipMalloc((void**)&WeightArrayDevice, rawMatrixSize);
	hipMalloc((void**)&CostArrayDevice, rawVertexSize);
	hipMalloc((void**)&UpdateCostArrayDevice, rawVertexSize);
	
	// MaskArray is used for determining that the vertex
	// is already been visited or not
	hipMalloc((void**)&MaskArrayDevice, rawVertexSize);
	
	MaskArrayHost[start] = 1;
	CostArrayHost[start] = 0;
	UpdateCostArrayHost[start] = 0;


	hipMemcpy(VertexArrayDevice, VertexArrayHost, 
			rawVertexSize, hipMemcpyHostToDevice);
	
	hipMemcpy(WeightArrayDevice, WeightArrayHost,
			rawMatrixSize, hipMemcpyHostToDevice);

	hipMemcpy(CostArrayDevice, CostArrayHost,
			rawVertexSize, hipMemcpyHostToDevice);
	
	hipMemcpy(UpdateCostArrayDevice, UpdateCostArrayHost,
			rawVertexSize, hipMemcpyHostToDevice);
	
	// freeing big memory that not been used
	// anymore because already copied to device
	free(UpdateCostArrayHost);
	free(WeightArrayHost);
	free(VertexArrayHost);
	
	
	
	while(!is_empty(MaskArrayHost, vertexSize * vertexSize)) {
		for(int ii = 0; ii < vertexSize; ii++) {
			
			first_cuda_ssp_kernel<<<gridDim, blockDim >>>(VertexArrayDevice, 
					WeightArrayDevice, MaskArrayDevice, CostArrayDevice,
					UpdateCostArrayDevice);

			second_cuda_ssp_kernel<<<gridDim, blockDim >>>(VertexArrayDevice,
					WeightArrayDevice, MaskArrayDevice, CostArrayDevice,
					UpdateCostArrayDevice);
		}

		// update the masks
		hipMemcpy(MaskArrayHost, MaskArrayDevice, 
				rawVertexSize, hipMemcpyDeviceToHost);
	}
	
	
	hipMemcpy(CostArrayHost, CostArrayDevice, rawVertexSize, 
			hipMemcpyDeviceToHost);
	
	
	int finalCost = 0;

	std::list<int> result = get_shortest_path(CostArrayHost, 
			start, end, vertexSize, finalCost);

	std::stringstream sstream(std::stringstream::in |
			std::stringstream::out);

	std::copy(result.begin(), result.end(), 
			std::ostream_iterator<g::vertex_t>(sstream, "->"));	
	
	std::string out;

	//while(!sstream.eof()) 
	//	sstream >> out;
	out = sstream.str();
	
	std::cout << "Path : \n" << out << "end" << std::endl;
	
	return 0;
}


std::list<int> get_shortest_path(float * CostArray, 
		const int source, const int target,
		const int vertexSize, int & finalCost) {
	std::list<int> result;
	
	
	return result;
}
