#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "defs.h"
#include "reader.h"

__device__ int thread_id() {
	return threadIdx.x;
}

__global__ void first_cuda_ssp_kernel() {
	
}	

__global__ void second_cuda_ssp_kernel() {
	
}


int main(int argc, char ** argv) {
	
	const char * filename = argv[1];

	std::adjmap graph;

	g::reader::read(filename, graph);

	int * VertexArray, * EdgeArray;
	float * WeightArray, * CostArray, * UpdateCostArray;
	bool * MaskArray;

	hipEvent_t start, stop;

	return 0;
}
